
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA() {
    printf("Hello from CUDA! Thread ID: %d\n", threadIdx.x);
}

int main() {
    helloCUDA<<<1, 5>>>();
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    } else {
        printf("Kernel executed successfully!\n");
    }
    return 0;
}